#include "hip/hip_runtime.h"
#include "kernel.cuh"

template <typename T> __global__ void sum_kernel(T *a, T *b, T *c, int n) {
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < n)
    c[id] = a[id] + b[id];
}

template <typename T> void wrap_sum_vector(T *a, T *b, T *res, int n) {
  int threads_per_block = 128;
  int blocks_per_grid = ceil(float(n) / threads_per_block);

  float *d_a, *d_b, *d_c;
  hipMalloc((void **)&d_a, n * sizeof(float));
  hipMalloc((void **)&d_b, n * sizeof(float));
  hipMalloc((void **)&d_c, n * sizeof(float));

  hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

  sum_kernel<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_c, n);

  hipMemcpy(res, d_c, n * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

template void wrap_sum_vector<int>(int *a, int *b, int *res, int n);
template void wrap_sum_vector<float>(float *a, float *b, float *res, int n);
template void wrap_sum_vector<double>(double *a, double *b, double *res, int n);
