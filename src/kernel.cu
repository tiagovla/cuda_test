#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "math.h"

__global__ void sum_kernel(float *a, float *b, float *c, int n) {
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < n)
    c[id] = a[id] + b[id];
}

void wrap_sum_vector(float *a, float *b, float *res, int n) {
  int threads_per_block = 256;
  int blocks_per_grid = ceil(float(n) / threads_per_block);

  float *d_a, *d_b, *d_c;
  hipMalloc((void **)&d_a, n * sizeof(float));
  hipMalloc((void **)&d_b, n * sizeof(float));
  hipMalloc((void **)&d_c, n * sizeof(float));

  hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

  sum_kernel<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_c, n);

  hipMemcpy(res, d_c, n * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
